#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <iomanip>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "Complex.h"

# define M_PI   3.14159265358979323846f
# define M_PI_2 1.57079632679489661923f
# define M_PI_4 0.785398163397448309616f
# define M_2_PI 0.636619772367581343076f
#define eps		1e-15f
#define el		0.5772156649015329f

#define N	5
#define NxN	(N * N)
#define H_T ((2.0f * M_PI) / (N - 1))

typedef complex<float> complexf;

__constant__ float a[14] = {
	-7.03125e-2,
	0.112152099609375,
	-0.5725014209747314,
	6.074042001273483,
	-1.100171402692467e2,
	3.038090510922384e3,
	-1.188384262567832e5,
	6.252951493434797e6,
	-4.259392165047669e8,
	3.646840080706556e10,
	-3.833534661393944e12,
	4.854014686852901e14,
	-7.286857349377656e16,
	1.279721941975975e19
};
__constant__ float b[14] = {
	7.32421875e-2,
	-0.2271080017089844,
	1.727727502584457,
	-2.438052969955606e1,
	5.513358961220206e2,
	-1.825775547429318e4,
	8.328593040162893e5,
	-5.006958953198893e7,
	3.836255180230433e9,
	-3.649010818849833e11,
	4.218971570284096e13,
	-5.827244631566907e15,
	9.476288099260110e17,
	-1.792162323051699e20
};

__device__ complexf dev_besselh01Function(complexf z)
{
	complexf z1, z2, cr, cp, cs, cp0, cq0, cp1, cq1, ct1, ct2, cu;
	complexf cj0, cy0;
	complexf cii(0.0, 1.0);
	complexf cone(1.0, 0.0);
	complexf czero(0.0, 0.0);

	float a0, w0;
	int k, kz;

	a0 = abs(z);
	z2 = z*z;
	z1 = z;
	if (a0 == 0.0f) {
		cj0 = cone;
		cy0 = complexf(-1e38f, 0.0f);
		return 0;
	}
	if (real(z) < 0.0f) z1 = -z;
	if (a0 <= 12.0f) {
		cj0 = cone;
		cr = cone;
		for (k = 1; k <= 40; k++) {
			cr *= -0.25f * z2 / (float)(k*k);
			cj0 += cr;
			if (abs(cr) < abs(cj0) * eps) break;
		}
		w0 = 0.0f;
		cr = cone;
		cs = czero;
		for (k = 1; k <= 40; k++) {
			w0 += 1.0f / k;
			cr *= -0.25f * z2 / (float)(k*k);
			cp = cr*w0;
			cs += cp;
			if (abs(cp) < abs(cs) * eps) break;
		}
		cy0 = M_2_PI * ((log(0.5f * z1) + el) * cj0 - cs);
	}
	else {
		if (a0 >= 50.0f) kz = 8;         // can be changed to 10
		else if (a0 >= 35.0f) kz = 10;   //   "      "     "  12
		else kz = 12;                    //   "      "     "  14
		ct1 = z1 - M_PI_4;
		cp0 = cone;
		for (k = 0; k<kz; k++) {
			cp0 += a[k] * pow(z1, -2.0f*k - 2.0f);
		}
		cq0 = -0.125f / z1;
		for (k = 0; k<kz; k++) {
			cq0 += b[k] * pow(z1, -2.0f*k - 3.0f);
		}
		cu = sqrt(M_2_PI / z1);
		cj0 = cu*(cp0*cos(ct1) - cq0*sin(ct1));
		cy0 = cu*(cp0*sin(ct1) + cq0*cos(ct1));
	}
	if (real(z) < 0.0f) {
		if (imag(z) < 0.0f) {
			cy0 -= 2.0f * cii * cj0;
		}
		else if (imag(z) > 0.0f) {
			cy0 += 2.0f * cii * cj0;
		}
	}
	return (cj0 + cii * cy0);
}

__host__ complexf host_besselh01Function(complexf z)
{
	complexf z1, z2, cr, cp, cs, cp0, cq0, cp1, cq1, ct1, ct2, cu;
	complexf cj0, cy0;
	complexf cii(0.0, 1.0);
	complexf cone(1.0, 0.0);
	complexf czero(0.0, 0.0);

	float a0, w0;
	int k, kz;

	a0 = abs(z);
	z2 = z*z;
	z1 = z;
	if (a0 == 0.0f) {
		cj0 = cone;
		cy0 = complexf(-1e38f, 0.0f);
		return 0;
	}
	if (real(z) < 0.0f) z1 = -z;
	if (a0 <= 12.0f) {
		cj0 = cone;
		cr = cone;
		for (k = 1; k <= 40; k++) {
			cr *= -0.25f * z2 / (float)(k*k);
			cj0 += cr;
			if (abs(cr) < abs(cj0) * eps) break;
		}
		w0 = 0.0f;
		cr = cone;
		cs = czero;
		for (k = 1; k <= 40; k++) {
			w0 += 1.0f / k;
			cr *= -0.25f * z2 / (float)(k*k);
			cp = cr*w0;
			cs += cp;
			if (abs(cp) < abs(cs) * eps) break;
		}
		cy0 = M_2_PI * ((log(0.5f * z1) + el) * cj0 - cs);
	}
	else {
		if (a0 >= 50.0f) kz = 8;         // can be changed to 10
		else if (a0 >= 35.0f) kz = 10;   //   "      "     "  12
		else kz = 12;                    //   "      "     "  14
		ct1 = z1 - M_PI_4;
		cp0 = cone;
		for (k = 0; k<kz; k++) {
			cp0 += a[k] * pow(z1, -2.0f*k - 2.0f);
		}
		cq0 = -0.125f / z1;
		for (k = 0; k<kz; k++) {
			cq0 += b[k] * pow(z1, -2.0f*k - 3.0f);
		}
		cu = sqrt(M_2_PI / z1);
		cj0 = cu*(cp0*cos(ct1) - cq0*sin(ct1));
		cy0 = cu*(cp0*sin(ct1) + cq0*cos(ct1));
	}
	if (real(z) < 0.0f) {
		if (imag(z) < 0.0f) {
			cy0 -= 2.0f * cii * cj0;
		}
		else if (imag(z) > 0.0f) {
			cy0 += 2.0f * cii * cj0;
		}
	}
	return (cj0 + cii * cy0);
}

__device__ float dev_computeR(int i, int j)
{
	float t_i = H_T * i;
	float t_j = H_T * j;
	float x1 = cos(t_i);
	float x2 = sin(t_i);
	float y1 = cos(t_j);
	float y2 = sin(t_j);
	return sqrtf(powf(x1 - y1, 2) + powf(x2 - y2, 2));
}

__host__ float host_computeR(int i, int j)
{
	float t_i = H_T * i;
	float t_j = H_T * j;
	float x1 = cos(t_i);
	float x2 = sin(t_i);
	float y1 = cos(t_j);
	float y2 = sin(t_j);
	return sqrtf(powf(x1 - y1, 2) + powf(x2 - y2, 2));
}

__global__ void complexKernel(complexf *result, float* rArray)
{
	complexf hi(1.7, -2.7);

	int mtxI = blockIdx.x;
	int mtxJ = threadIdx.x;

	float R = dev_computeR(mtxI, mtxJ);

	int mtxIdx = mtxI * N + mtxJ;

	rArray[mtxIdx] = R;

	result[mtxIdx] = dev_besselh01Function(hi * R);
}

__host__ void printArray(float *array, unsigned size) {
	for (unsigned i = 0; i < size; ++i) {
		printf(" [%d]: %f; ", i, array[i]);
	}
}

__host__ void printArray(complexf *array, unsigned size) {
	for (unsigned i = 0; i < size; ++i) {
		printf(" [%d]: ", i);
		std::cout << array[i];
	}
}

__host__ complexf computeSummError(complexf *dev_besselResults) {
	complexf hi(1.7, -2.7);
	complexf result(0.0f, 0.0f);
	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			float R = host_computeR(i, j);
			int mtxIdx = i * N + j;
			result += host_besselh01Function(hi * R) - dev_besselResults[mtxIdx];
		}
	}
	return result;
}

int main()
{
	complexf *dev_result;
	hipMalloc((void**)&dev_result, sizeof(complexf) * NxN);

	float *dev_rArray;
	hipMalloc((void**)&dev_rArray, sizeof(float) * NxN);

	complexKernel << <N, N >> >(dev_result, dev_rArray);

	complexf *result = new complexf[NxN];
	float *rArray = new float[NxN];

	hipMemcpy(result, dev_result, sizeof(complexf)* NxN, hipMemcpyDeviceToHost);
	hipMemcpy(rArray, dev_rArray, sizeof(float)* NxN, hipMemcpyDeviceToHost);

	printf("N: %d ", N);

	/*printf("\n\n\n R: \n");
	printArray(rArray, NxN);

	printf("\n\n\n Rsutlt: \n");
	printArray(result, NxN);*/

	std::cout << "\n\n Result [2]: " << rArray[2];
	std::cout << "\n\n Result [32]: " << rArray[32];
	std::cout << "\n\n Result [128]: " << rArray[128];

	std::cout << "\n\n Summary error: " << computeSummError(result);

	hipFree(&dev_rArray);
	hipFree(&dev_result);

	delete rArray;
	delete result;

	getchar();

	return 0;
}

