#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <iomanip>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "Complex.h"

# define M_PI   3.14159265358979323846f
# define M_PI_2 1.57079632679489661923f
# define M_PI_4 0.785398163397448309616f
# define M_2_PI 0.636619772367581343076f
#define eps		1e-15f
#define el		0.5772156649015329f

#define N	5
#define H_T ((2.0f * M_PI) / (N - 1))

typedef complex<float> complexf;

__constant__ float a[14] = {
	-7.03125e-2,
	0.112152099609375,
	-0.5725014209747314,
	6.074042001273483,
	-1.100171402692467e2,
	3.038090510922384e3,
	-1.188384262567832e5,
	6.252951493434797e6,
	-4.259392165047669e8,
	3.646840080706556e10,
	-3.833534661393944e12,
	4.854014686852901e14,
	-7.286857349377656e16,
	1.279721941975975e19 
};
__constant__ float b[14] = {
	7.32421875e-2,
	-0.2271080017089844,
	1.727727502584457,
	-2.438052969955606e1,
	5.513358961220206e2,
	-1.825775547429318e4,
	8.328593040162893e5,
	-5.006958953198893e7,
	3.836255180230433e9,
	-3.649010818849833e11,
	4.218971570284096e13,
	-5.827244631566907e15,
	9.476288099260110e17,
	-1.792162323051699e20 
};

__device__ complexf besselh01Function(complexf z)
{
	complexf z1, z2, cr, cp, cs, cp0, cq0, cp1, cq1, ct1, ct2, cu;
	complexf cj0, cy0;
	complexf cii(0.0, 1.0);
	complexf cone(1.0, 0.0);
	complexf czero(0.0, 0.0);

	float a0, w0;
	int k, kz;

 	a0 = abs(z);
	z2 = z*z;
	z1 = z;
	if (a0 == 0.0f) {
		cj0 = cone;
		cy0 = complexf(-1e38f, 0.0f);
		return 0;
	}
	if (real(z) < 0.0f) z1 = -z;
	if (a0 <= 12.0f) {
		cj0 = cone;
		cr = cone;
		for (k = 1; k <= 40; k++) {
			cr *= -0.25f * z2 / (float) (k*k);
			cj0 += cr;
			if (abs(cr) < abs(cj0) * eps) break;
		}
		w0 = 0.0f;
		cr = cone;
		cs = czero;
		for (k = 1; k <= 40; k++) {
			w0 += 1.0f / k;
			cr *= -0.25f * z2 / (float) (k*k);
			cp = cr*w0;
			cs += cp;
			if (abs(cp) < abs(cs) * eps) break;
		}
		cy0 = M_2_PI * ((log(0.5f * z1) + el) * cj0 - cs);
	}
	else {
		if (a0 >= 50.0f) kz = 8;         // can be changed to 10
		else if (a0 >= 35.0f) kz = 10;   //   "      "     "  12
		else kz = 12;                    //   "      "     "  14
		ct1 = z1 - M_PI_4;
		cp0 = cone;
		for (k = 0; k<kz; k++) {
			cp0 += a[k] * pow(z1, -2.0f*k - 2.0f);
		}
		cq0 = -0.125f / z1;
		for (k = 0; k<kz; k++) {
			cq0 += b[k] * pow(z1, -2.0f*k - 3.0f);
		}
		cu = sqrt(M_2_PI / z1);
		cj0 = cu*(cp0*cos(ct1) - cq0*sin(ct1));
		cy0 = cu*(cp0*sin(ct1) + cq0*cos(ct1));
	}
	if (real(z) < 0.0f) {
		if (imag(z) < 0.0f) {
			cy0 -= 2.0f * cii * cj0;
		}
		else if (imag(z) > 0.0f) {
			cy0 += 2.0f * cii * cj0;
		}
	}
	return (cj0 + cii * cy0);
}

__device__ float computeR(int i, int j) 
{
	float t_i = H_T * i;
	float t_j = H_T * j;
	float x1 = cos(t_i);
	float x2 = sin(t_i);
	float y1 = cos(t_j);
	float y2 = sin(t_j);
	return sqrtf(powf(x1 - y1, 2) + powf(x2 - y2, 2));
}

__global__ void complexKernel(complexf *result, float* rArray)
{
	complexf hi(1.7, -2.7);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int elemsBeneathDiag = 0;
	for (int i = N, negateDispl = idx - i; negateDispl >= 0; i--) {
		elemsBeneathDiag += elemsBeneathDiag + 1;
		negateDispl -= i;
	}
	int displacedIdx = idx + elemsBeneathDiag;
	

	int mtxI = displacedIdx / N;
	int mtxJ = displacedIdx % N;

	float R = computeR(mtxI, mtxJ);

	int mtxIdx = mtxI * N + mtxJ;
	int transposeMtxIdx = mtxJ * N + mtxI;

	rArray[mtxIdx] = R;
	rArray[transposeMtxIdx] = R;

	result[mtxIdx] = besselh01Function(hi * R);
	result[transposeMtxIdx] = result[mtxIdx];
}

__host__ void printArray(float *array, unsigned size) {
	for (unsigned i = 0; i < size; ++i) {
		printf(" [%d]: %f; ", i, array[i]);
	}
}

__host__ void printArray(complexf *array, unsigned size) {
	for (unsigned i = 0; i < size; ++i) {
		printf(" [%d]: ", i);
		std::cout << array[i];
	}
}

int main()
{
	int elemsCnt  = N * N;
	int diagonalMtxElemsCount = (elemsCnt - N) / 2 + N;

	printf("\n\n diagonalElemsCount = %d \n", diagonalMtxElemsCount);

	dim3 threadsPerBlock(2); // TODO: Make 32 and higher
	dim3 numBlocks((diagonalMtxElemsCount + threadsPerBlock.x - 1) / threadsPerBlock.x);

	printf("\n\n num blocks = %d \n", numBlocks.x);

	int extendedElemsCount = N * N + threadsPerBlock.x - 1;

	printf("\n\n num extendedElemsCount = %d \n", extendedElemsCount);

	complexf *dev_result;
	hipMalloc((void**)&dev_result, sizeof(complexf) * extendedElemsCount);

	float *dev_rArray;
	hipMalloc((void**)&dev_rArray, sizeof(float) * extendedElemsCount);

	complexKernel << <numBlocks, threadsPerBlock >> >(dev_result, dev_rArray);

	complexf *result = new complexf[N * N];
	float *rArray = new float[N * N];

	hipMemcpy(result, dev_result, sizeof(complexf) * N * N, hipMemcpyDeviceToHost);
	hipMemcpy(rArray, dev_rArray, sizeof(float) * N * N, hipMemcpyDeviceToHost);

	printf("N: %d \n\n\n", N);

	printf("R: \n");
	printArray(rArray, N * N);

	/*printf("\n\n\n Rsutlt: \n");
	printArray(result, N * N);*/

	hipFree(&dev_rArray);
	hipFree(&dev_result);

	delete rArray;
	delete result;

	getchar();

    return 0;
}

