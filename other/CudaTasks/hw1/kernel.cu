#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>

#include "Console.h"

#define N 16
#define M 8
#define BLOCKS 10
#define THREADS 64

#define OUT_OF_ARRAY -18

using namespace std;
using namespace utils;

__global__ void kernel_task1(int *vecBlockIdx, int *vecThreadIdx) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		vecBlockIdx[idx] = blockIdx.x;
		vecThreadIdx[idx] = threadIdx.x;
	}
}

void task1() {
	int *dev_vecBlockIdx = 0;
	int *dev_vecThreadIdx = 0;
	hipMalloc((void**)&dev_vecBlockIdx, N * sizeof(int));
	hipMalloc((void**)&dev_vecThreadIdx, N * sizeof(int));

	kernel_task1 << <N / THREADS + 1, THREADS >> >(dev_vecBlockIdx, dev_vecThreadIdx);

	int *vecBlockIdx = new int[N];
	int *vecThreadIdx = new int[N];

	hipMemcpy(vecBlockIdx, dev_vecBlockIdx, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(vecThreadIdx, dev_vecThreadIdx, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_vecBlockIdx);
	hipFree(dev_vecThreadIdx);

	printf("\n\nResults on host.");

	printf("\n\nvecBlockIdx: \n");
	// ����� ��� ������� ������ ��� ������� �����������
	Console::printMatrix(vecBlockIdx, 16, 8);

	printf("\n\nvecThreadIdx: \n");
	Console::printMatrix(vecThreadIdx, 16, 8);

	getchar();

	delete[] vecBlockIdx;
	delete[] vecThreadIdx;
}


__global__ void kernel_task2(int *matBlockIdx, int *matThreadIdx, int blocksCount, int threadsCount) {
	int _blockIdx = blockIdx.x * gridDim.y + blockIdx.y;

	if (_blockIdx < blocksCount) {
		matBlockIdx[_blockIdx] = blockIdx.x;
	} else { 
		matBlockIdx[_blockIdx] = OUT_OF_ARRAY;
	}

	int _threadIdxI = blockIdx.x * blockDim.x + threadIdx.x;
	int _threadIdxJ = blockIdx.y * blockDim.y + threadIdx.y;
	int _threadIdx = _threadIdxI * gridDim.y * blockDim.x + _threadIdxJ;
	if (_threadIdx < threadsCount) {
		matThreadIdx[_threadIdx] = threadIdx.x;
	} else {
		matThreadIdx[_threadIdx] = OUT_OF_ARRAY;
	}
}

void task2() {

	dim3 threadsPerBlock(8, 4);
	dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
	unsigned blocksCount = numBlocks.x * numBlocks.y;
	unsigned threadsCount = numBlocks.x * threadsPerBlock.x * numBlocks.y * threadsPerBlock.y;

	cout << blocksCount << " " << threadsCount << endl;

	int *dev_matBlockIdx = 0;
	int *dev_matThreadIdx = 0;
	hipMalloc((void**)&dev_matBlockIdx, blocksCount * sizeof(int));
	hipMalloc((void**)&dev_matThreadIdx, threadsCount * sizeof(int));

	kernel_task2 << <numBlocks, threadsPerBlock >> >(dev_matBlockIdx, dev_matThreadIdx, blocksCount, threadsCount);

	int *matBlockIdx = new int[blocksCount];
	int *matThreadIdx = new int[threadsCount];

	hipMemcpy(matBlockIdx, dev_matBlockIdx, blocksCount * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(matThreadIdx, dev_matThreadIdx, threadsCount * sizeof(int), hipMemcpyDeviceToHost);

	printf("\n\nResults on host.");

	printf("\n\nmatBlockIdx: \n");
	Console::printMatrix(matBlockIdx, numBlocks.x, numBlocks.y);

	printf("\n\nmatThreadIdx: \n");
	Console::printMatrix(matThreadIdx, numBlocks.x * threadsPerBlock.x, numBlocks.y * threadsPerBlock.y);

	getchar();

	hipFree(dev_matBlockIdx);
	hipFree(dev_matThreadIdx);

	delete[] matBlockIdx;
	delete[] matThreadIdx;
}

int* generateArray(int size, int dispersion) {
	int* array = new int[size];
	for (int i = 0; i < size; ++i) {
		array[i] = rand() % dispersion;
	}
	return array;
}

__global__ void kernel_task3(int *matrix, int *vector, int *outVector, int rows, int cols) {
	int firstRowElemIdx = blockIdx.x * gridDim.y * blockDim.x;
	if (firstRowElemIdx < rows * cols) {
		int sum = 0;
		for (size_t col = 0; col < cols; col++) {
			sum += matrix[firstRowElemIdx + col] * vector[col];
		}
		outVector[blockIdx.x] = sum;
	}
}

void task3() {
	int rows = 3;
	int cols = 3;
	int elemsCount = rows * cols;

	int threadsPerBlock = rows;
	int blocksCount = cols / threadsPerBlock + 1;

	int *matrix = generateArray(elemsCount, 3);
	int *vector = generateArray(cols, 3);

	printf("\n Generated matrix: \n");
	Console::printMatrix(matrix, rows, cols);

	printf("\n Generated vector: ");
	Console::printArray(vector, cols);

	int *dev_matrix = 0;
	int *dev_vector = 0;
	int *dev_outVector = 0;
	hipMalloc((void**)&dev_matrix, elemsCount * sizeof(int));
	hipMalloc((void**)&dev_vector, cols * sizeof(int));
	hipMalloc((void**)&dev_outVector, cols * sizeof(int));

	hipMemcpy(dev_matrix, matrix, elemsCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_vector, vector, cols * sizeof(int), hipMemcpyHostToDevice);

	kernel_task3 << <blocksCount, threadsPerBlock >> > (dev_matrix, dev_vector, dev_outVector, rows, cols);

	int *outVector = new int[cols];

	hipMemcpy(outVector, dev_outVector, cols * sizeof(int), hipMemcpyDeviceToHost);

	printf("\n\n Result vector: ");
	Console::printArray(outVector, cols);

	getchar();

	hipFree(dev_matrix);
	hipFree(dev_vector);
	hipFree(dev_outVector);

	delete[] matrix;
	delete[] vector;
	delete[] outVector;
}

#define M_PI 3.14159265358979323846 
#define T 2 * M_PI
#define EPS 0.0001

float* arrayFromSegment(float pointA, float pointB, unsigned partsCount) {
	float* array = new float[partsCount];
	float step = (pointB - pointA) / (partsCount - 1);
	for (int i = 0; i < partsCount; ++i) {
		array[i] = pointA + step * i;
	}
	return array;
}

#define ROWS 10
#define COLS 5
#define ELEMS_COUNT (ROWS * COLS)

__global__ void kernel_task4A(float *matrix, float *vectorX, float *vectorY) {
	int _threadIdxI = blockIdx.x * blockDim.x + threadIdx.x;
	int _threadIdxJ = blockIdx.y * blockDim.y + threadIdx.y;
	int _threadIdx = _threadIdxI * gridDim.y * blockDim.x + _threadIdxJ;

	int mtxIdxI = _threadIdx / COLS;
	int mtxIdxJ = _threadIdx - (mtxIdxI * COLS);
	
	matrix[_threadIdx] = T * __sinf(vectorX[mtxIdxI]) * __cosf(vectorY[mtxIdxJ]);
}

void task4A() {
	printf("Rows count: %d, Cols count: %d \n", ROWS, COLS);

	dim3 threadsPerBlock(2, 2);
	dim3 blocksCount((ROWS + threadsPerBlock.x - 1) / threadsPerBlock.x,
					 (COLS + threadsPerBlock.y - 1) / threadsPerBlock.y);

	float *vectorX = arrayFromSegment(1.0f, 5.0f, COLS);
	float *vectorY = arrayFromSegment(1.0f, 5.0f, ROWS);

	printf("\n Generated vector X: \n");
	Console::printArray(vectorX, COLS);

	printf("\n Generated vector Y: \n");
	Console::printArray(vectorY, ROWS);

	float *dev_matrix = 0;
	float *dev_vectorX = 0;
	float *dev_vectorY = 0;
	hipMalloc((void**)&dev_matrix, ELEMS_COUNT * sizeof(float));
	hipMalloc((void**)&dev_vectorX, COLS * sizeof(float));
	hipMalloc((void**)&dev_vectorY, ROWS * sizeof(float));

	hipMemcpy(dev_vectorX, vectorX, COLS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vectorY, vectorY, ROWS * sizeof(float), hipMemcpyHostToDevice);

	kernel_task4A << <blocksCount, threadsPerBlock >> > (dev_matrix, dev_vectorX, dev_vectorY);

	float *matrix = new float[ELEMS_COUNT];

	hipMemcpy(matrix, dev_matrix, ELEMS_COUNT * sizeof(float), hipMemcpyDeviceToHost);

	printf("\n\n Result matrix: \n");
	Console::printMatrix(matrix, ROWS, COLS);

	getchar();

	hipFree(dev_matrix);
	hipFree(dev_vectorX);
	hipFree(dev_vectorY);

	delete[] matrix;
	delete[] vectorX;
	delete[] vectorY;
}

__device__ float numberFromSegment(float pointA, float pointB, unsigned partsCount, unsigned currStep) {
	float step = (pointB - pointA) / (partsCount - 1);
	return pointA + step * currStep;
}

__global__ void kernel_task4B(float *matrix, float s1PointA, float s1PointB, float s2PointA, float s2PointB) {
	int _threadIdxI = blockIdx.x * blockDim.x + threadIdx.x;
	int _threadIdxJ = blockIdx.y * blockDim.y + threadIdx.y;
	int _threadIdx = _threadIdxI * gridDim.y * blockDim.x + _threadIdxJ;

	int mtxIdxI = _threadIdx / COLS;
	int mtxIdxJ = _threadIdx - (mtxIdxI * COLS);

	float x = numberFromSegment(s1PointA, s1PointB, COLS, mtxIdxJ);
	float y = numberFromSegment(s2PointA, s2PointB, ROWS, mtxIdxI);

	matrix[_threadIdx] = T * __sinf(x) * __cosf(y);
}


void task4BC() {
	printf("Rows count: %d, Cols count: %d \n", ROWS, COLS);

	dim3 threadsPerBlock(2, 2);
	dim3 blocksCount((ROWS + threadsPerBlock.x - 1) / threadsPerBlock.x,
					 (COLS + threadsPerBlock.y - 1) / threadsPerBlock.y);

	float segment1PointA = 1.0;
	float segment1PointB = 5.0;
	float segment2PointA = 3.0;
	float segment2PointB = 15.0;

	float *dev_matrix = 0;
	hipMalloc((void**)&dev_matrix, ELEMS_COUNT * sizeof(float));

	kernel_task4B << <blocksCount, threadsPerBlock >> > (dev_matrix, segment1PointA, segment1PointB, segment2PointA, segment2PointB);

	float *fromDev_matrix = new float[ELEMS_COUNT];
	hipMemcpy(fromDev_matrix, dev_matrix, ELEMS_COUNT * sizeof(float), hipMemcpyDeviceToHost);

	printf("\n\n Result from device matrix: \n");
	// Console::printMatrix(fromDev_matrix, ROWS, COLS);

	float *vectorX = arrayFromSegment(segment1PointA, segment1PointB, COLS);
	float *vectorY = arrayFromSegment(segment2PointA, segment2PointB, ROWS);
	float *host_matrix = new float[ELEMS_COUNT];
	float totalError = 0.;
	for (size_t i = 0; i < ROWS; i++) {
		for (size_t j = 0; j < COLS; j++) {
			host_matrix[i * COLS + j] = T * sinf(vectorX[j]) * cosf(vectorY[i]);
			totalError += host_matrix[i * COLS + j] - fromDev_matrix[i * COLS + j];
		}
	}

	printf("\n\n Result from host matrix: \n");
	// Console::printMatrix(host_matrix, ROWS, COLS);

	printf("\n\n Total error: %f", totalError);

	getchar();

	hipFree(dev_matrix);

	delete[] vectorX;
	delete[] vectorY;
	delete[] fromDev_matrix;
	delete[] host_matrix;
}


int main() {

	// task1();
	// task2();
	// task3();
	task4A();
	// task4BC();

	return 0;
}